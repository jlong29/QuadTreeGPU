#include "hip/hip_runtime.h"
//STL
#include <stdio.h>
#include <iostream>
#include <string.h>
#include <time.h>

//CUDA
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_functions.h>    // sdkCreateTimer: contains definitions in header

//OPENGL
#include <helper_gl.h>
#include <GL/freeglut.h>

//CUDA OpenGL interoperability
#include <cuda_gl_interop.h>
#define REFRESH_DELAY     10 //ms

#include "kernels.cuh"

//Reset Quad Tree state with new data
static bool bReset      = false;

// GLOBAL Variables //
//Problem Size
//Parameters
int N;
int W;
int H;

int window_width;
int window_height;
float aspRat;

//CUDA
float*   h_noiseX;
float*   h_noiseY;
float*   d_noiseX;
float*   d_noiseY;
size_t noiseSz;

uchar4* d_img;
size_t imgSz;

float elpsTime;
hipEvent_t start, stop;

//OPENGL
//Texture variables
GLuint imageTex;
struct hipGraphicsResource *pcuImageRes;

// Timing Code
//OpenGL loop = GPU + host
StopWatchInterface *timer;
int fpsCount;        // FPS count for averaging
int fpsLimit;        // FPS limit for sampling
float avgFPS;
uint frameCount;

/////////////////////////////////
// OPENGL FORWARD DECLARATIONS //
/////////////////////////////////
bool initGL(int *argc, char **argv);

//IMAGE DATA
void display();
void keyboard(unsigned char key, int x, int y);
void cleanup();
void timerEvent(int value);

#include "kernels.cuh"

static inline int divUp(int x, int y)
{
	return (x + y - 1) / y;
}

int main(int argc, char** argv)
{
	//Input Parameters
	N = 16;
	W = 640;
	H = 480;

	window_width  = W;
	window_height = H;

	//Set buffer pointers to NULL
	h_noiseX = NULL;
	h_noiseY = NULL;
	d_noiseX = NULL;
	d_noiseY = NULL;
	noiseSz =  N*sizeof(float);

	d_img   = NULL;
	imgSz   = W*H*sizeof(uchar4);

	//Root GPU Launch Optimization
	//1D
	int threads = NTHREADS;
	int blocks  = divUp(N, NTHREADS);

	//2D
	dim3 blockDim, gridDim;
	blockDim.x = WARPSIZE;
	blockDim.y = NWARPS;

	gridDim.x  = divUp(W, blockDim.x);
	gridDim.y  = divUp(H, blockDim.y);

	//initialize timers
	fpsCount   = 0;
	fpsLimit   = 1;
	avgFPS     = 0.0f;
	frameCount = 0;

	/* Set Up */
	//Set Device
	int deviceCount;
	checkCudaErrors(hipGetDeviceCount(&deviceCount));
	if (deviceCount == 0) {
		printf("There is no device supporting CUDA\n");
		return EXIT_FAILURE;
	}

	int dev = 0;
	if (dev >= deviceCount){
		printf("Input error: dev >= deviceCount\n");
		return EXIT_FAILURE;
	}

	hipSetDevice(dev);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);

	printf("\tDevice %d: \"%s\"\n", dev, deviceProp.name);

	// OpenGL: initialize on this device and set up windows
	if (false == initGL(&argc, argv))
	{
		return -1;
	}

	// Create the CUTIL timer
	sdkCreateTimer(&timer);

	//Allocate Memory
	checkCudaErrors(hipHostMalloc((void **)&h_noiseX, noiseSz));
	checkCudaErrors(hipHostMalloc((void **)&h_noiseY, noiseSz));
	checkCudaErrors(hipMalloc((void **)&d_noiseX, noiseSz));
	checkCudaErrors(hipMalloc((void **)&d_noiseY, noiseSz));
	checkCudaErrors(hipMalloc((void **)&d_img, imgSz));

	//Set Timers
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	
	//Generate Random Data
	checkCudaErrors(hipEventRecord(start, 0));

	int seed = (int)time(0);

	generate_uniform2D_kernel<<<blocks, threads>>>(d_noiseX, d_noiseY, seed, W, H, N);

	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elpsTime, start, stop));
	printf("\n\nElapsed time for random number generation:  %9.6f ms \n", elpsTime);
	
	//Set Image to Black
	d_setBlackImag<<<gridDim, blockDim>>>(d_img, W, H);
	// checkCudaErrors(hipMemset(d_img, 0, sizeof(uchar4)*W*H));

	//Write Random data onto image buffer
	checkCudaErrors(hipEventRecord(start, 0));

	d_writeData2Image<<<blocks, threads>>>(d_img, d_noiseX, d_noiseY, W, H, N);

	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elpsTime, start, stop));
	printf("\n\nElapsed time for writing noise data:        %9.6f ms \n", elpsTime);

	//Copy back to host for checking    
	checkCudaErrors(hipMemcpy(h_noiseX, d_noiseX, noiseSz, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_noiseY, d_noiseY, noiseSz, hipMemcpyDeviceToHost));

	printf("Printing out 2D Noise:\n\t");
	for (int ii = 0; ii < min(100, N); ii++)
	{
		printf("[%d, %d], ", (int)h_noiseX[ii], (int)h_noiseY[ii]);
	}
	printf("\n");

	//convert device memory to texture
	hipArray_t ArrIm;
	hipGraphicsMapResources(1, &pcuImageRes, 0);
	hipGraphicsSubResourceGetMappedArray(&ArrIm, pcuImageRes, 0, 0);

	checkCudaErrors(hipMemcpyToArray(ArrIm, 0, 0, d_img, imgSz, hipMemcpyDeviceToDevice));
	hipGraphicsUnmapResources(1, &pcuImageRes, 0);

	////////////////////////////////
	// LAUNCH OPENGL DISPLAY LOOP //
	////////////////////////////////
	glutMainLoop();
	return 0;
}

//OpenGL function definitions
void computeFPS()
{
	frameCount++;
	fpsCount++;

	if (fpsCount == fpsLimit)
	{
		avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
		fpsCount = 0;
		fpsLimit = (int)MAX(avgFPS, 1.f);

		sdkResetTimer(&timer);
	}
	
	char fps[256];
	sprintf(fps, "Quad Tree: %3.1f fps", avgFPS);
	glutSetWindowTitle(fps);
}

bool initGL(int *argc, char **argv)
{
	glutInit(argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);

	//Create Image Data Window
	glutInitWindowSize(window_width, window_height);
	glutCreateWindow("Quad Tree");
	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	glutReportErrors();

	// NOTE: These calls don't work until first window is created

	//initialize necessary OpenGL extensions
	if (!isGLVersionSupported(2,0))
	{
		fprintf(stderr,"ERROR: Support for necessary OpenGL extensions missing.");
		fflush(stderr);
		return false;
	}
	fprintf(stdout,"OpenGL version supported by this platform: (%s)\n", glGetString(GL_VERSION));

	// Create and Register OpenGL Texture for Image(1 channel)
	hipError_t err1;
	glGenTextures(1, &imageTex);
	glBindTexture(GL_TEXTURE_2D, imageTex);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, window_width, window_height, 0, GL_BGRA, GL_UNSIGNED_BYTE, NULL);
	glBindTexture(GL_TEXTURE_2D, 0);

	err1 = hipGraphicsGLRegisterImage(&pcuImageRes, imageTex, GL_TEXTURE_2D, cudaGraphicsMapFlagsWriteDiscard);

	if (err1 != 0)
	{
		fprintf(stderr,"ERROR: Registering openGL texture failed\n");
		return false;
	}
	glutReportErrors();

	// Image Data: register callbacks
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutTimerFunc(REFRESH_DELAY, timerEvent, 0);
	glutCloseFunc(cleanup);

	return true;
}

void display()
{
	//Time it
	sdkStartTimer(&timer);

	if (bReset)
	{
		bReset = false;

		//Root GPU Launch Optimization
		//1D
		int threads = NTHREADS;
		int blocks  = divUp(N, NTHREADS);

		//2D
		dim3 blockDim, gridDim;
		blockDim.x = WARPSIZE;
		blockDim.y = NWARPS;

		gridDim.x  = divUp(W, blockDim.x);
		gridDim.y  = divUp(H, blockDim.y);

		//Generate Random Data
		checkCudaErrors(hipEventRecord(start, 0));

		int seed = (int)time(0);

		generate_uniform2D_kernel<<<blocks, threads>>>(d_noiseX, d_noiseY, seed, W, H, N);

		checkCudaErrors(hipEventRecord(stop, 0));
		checkCudaErrors(hipEventSynchronize(stop));
		checkCudaErrors(hipEventElapsedTime(&elpsTime, start, stop));
		printf("\n\nElapsed time for random number generation:  %9.6f ms \n", elpsTime);
		
		//Set Image to Black
		d_setBlackImag<<<gridDim, blockDim>>>(d_img, W, H);
		// checkCudaErrors(hipMemset(d_img, 0, sizeof(uchar4)*W*H));

		//Write Random data onto image buffer
		checkCudaErrors(hipEventRecord(start, 0));

		d_writeData2Image<<<blocks, threads>>>(d_img, d_noiseX, d_noiseY, W, H, N);

		checkCudaErrors(hipEventRecord(stop, 0));
		checkCudaErrors(hipEventSynchronize(stop));
		checkCudaErrors(hipEventElapsedTime(&elpsTime, start, stop));
		printf("\n\nElapsed time for writing noise data:        %9.6f ms \n", elpsTime);

		//Copy back to host for checking    
		checkCudaErrors(hipMemcpy(h_noiseX, d_noiseX, noiseSz, hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(h_noiseY, d_noiseY, noiseSz, hipMemcpyDeviceToHost));

		printf("Printing out 2D Noise:\n\t");
		for (int ii = 0; ii < min(100, N); ii++)
		{
			printf("[%d, %d], ", (int)h_noiseX[ii], (int)h_noiseY[ii]);
		}
		printf("\n");

		//convert device memory to texture
		hipArray_t ArrIm;
		hipGraphicsMapResources(1, &pcuImageRes, 0);
		hipGraphicsSubResourceGetMappedArray(&ArrIm, pcuImageRes, 0, 0);

		//NOTE: DISTORTION MODEL: 3 * d_dstReSz
		checkCudaErrors(hipMemcpyToArray(ArrIm, 0, 0, d_img, imgSz, hipMemcpyDeviceToDevice));
		hipGraphicsUnmapResources(1, &pcuImageRes, 0);
	}

	/////////////////////////
	// DISPLAY WITH OPENGL //
	/////////////////////////

	//OpenGL Part
	glClear(GL_COLOR_BUFFER_BIT);
	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);

	glBindTexture(GL_TEXTURE_2D, imageTex);
	glEnable(GL_TEXTURE_2D);
	glDisable(GL_DEPTH_TEST);
	glDisable(GL_LIGHTING);
	glTexEnvf(GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_REPLACE);

	//Note: texCoords go clockwise from top left and verts go counter-clockwise from lower left
	glBegin(GL_QUADS);
	glTexCoord2f(0.0, 1.0);
	glVertex3f(-1, -1.0, 0.0);
	glTexCoord2f(1.0, 1.0);
	glVertex3f(1, -1.0, 0.0);
	glTexCoord2f(1.0, 0.0);
	glVertex3f(1, 1.0, 0.0);
	glTexCoord2f(0.0, 0.0);
	glVertex3f(-1, 1.0, 0.0);
	glEnd();
	glBindTexture(GL_TEXTURE_2D, 0);
	glDisable(GL_TEXTURE_2D);

	glutReportErrors();

	hipDeviceSynchronize();

	// Updating timing information
	sdkStopTimer(&timer);
	computeFPS();

	//swap
	glutSwapBuffers();
}

//Keyboard callback
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
	fprintf(stdout,"\tKey Press: %u\n", (uint)key);
	switch (key)
	{
	case (27) :
	{
		glutDestroyWindow(glutGetWindow());
		return;
	}
	case 'r':
	{
		//r: Reset Quad Tree with new data
		bReset = true;
		return;
	}
	}
}

void timerEvent(int value)
{
	if (glutGetWindow())
	{
		glutPostRedisplay();
		glutTimerFunc(REFRESH_DELAY, timerEvent,0);
	}
}

void cleanup()
{
	sdkDeleteTimer(&timer);

	// CUDA/OPENGL
	fprintf(stdout,"\tCUDA:\n");
	hipGraphicsUnregisterResource(pcuImageRes);
	glDeleteTextures(1, &imageTex);
	imageTex = 0;
	fprintf(stdout,"\t\tAll openGL resources cleaned\n");

	//Deallocate memory
	checkCudaErrors(hipHostFree(h_noiseX));
	checkCudaErrors(hipHostFree(h_noiseY));
	checkCudaErrors(hipFree(d_noiseX));
	checkCudaErrors(hipFree(d_noiseY));
	checkCudaErrors(hipFree(d_img));

	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));
	fprintf(stdout,"\t\tAll Cuda resources cleaned\n");
}
