#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_math_constants.h>
#include <hiprand/hiprand_kernel.h>

#include "quadTreeKernels.h"

namespace quadTreeKernels
{

#define DEBUG

// Image Processing //
//Image bounds check
__device__ __forceinline__ int in_img(int x, int y, int w, int h)
{
	return x >= 0 && x < w && y >= 0 && y < h;
}
//pixel coloring
__device__ __inline__ void setRedHue(const uchar& hue, uchar4& RGBA)
{
	RGBA.x = hue;
	RGBA.y = 0;
	RGBA.z = 0;
	RGBA.w = 255;
}
__device__ __inline__ void setGreenHue(const uchar& hue, uchar4& RGBA)
{
	RGBA.x = 0;
	RGBA.y = hue;
	RGBA.z = 0;
	RGBA.w = 255;

}
__device__ __inline__ void setBlueHue(const uchar& hue, uchar4& RGBA)
{
	RGBA.x = 0;
	RGBA.y = 0;
	RGBA.z = hue;
	RGBA.w = 255;
}
__device__ __inline__ void setBlack(uchar4& RGBA)
{
	// points as black(transparent)
	RGBA.x = 0;
	RGBA.y = 0;
	RGBA.z = 0;
	RGBA.w = 0;
}

//Black Image: device and host code
__global__ void d_setBlackImag(uchar4* dst, const int w, const int h)
{
	//Position of the thread in the image
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	//Early exit if the thread is not in the image
	if (!in_img(x, y, w, h))
		return;

	//Set to RGBA black
	// setGreenHue(255, dst[y*w + x]);
	setBlack(dst[y*w + x]);
}

//Write Random data onto image buffer
__global__ void d_writeData2Image(uchar4* dst, const float* __restrict noiseX, const float* __restrict noiseY,const int w, const int h, const int n)
{
	int idx        = threadIdx.x + blockIdx.x * blockDim.x;
	int numThreads = blockDim.x*gridDim.x;

	for(int i = idx; i < n; i+=numThreads)
	{
		int shotX = (int)noiseX[i];
		int shotY = (int)noiseY[i];
		if (in_img(shotX, shotY, w, h))
			setGreenHue(255, dst[shotY*w + shotX]);
	}
}

__global__ void d_writeFilter2Image(uchar4* dst, const float* __restrict filterX, const float* __restrict filterY,const int w, const int h, const int n)
{
	int idx        = threadIdx.x + blockIdx.x * blockDim.x;
	int numThreads = blockDim.x*gridDim.x;

	for(int i = idx; i < n; i+=numThreads)
	{
		int shotX = (int)filterX[i];
		int shotY = (int)filterY[i];
		if (in_img(shotX, shotY, w, h))
		{
			setBlueHue(255, dst[shotY*w + shotX]);
		}
	}
}

//Draw internal edges of cells
__global__ void d_drawCellInnerEdges(uchar4* dst, int* index, const float* __restrict x, const float* __restrict y, const float* __restrict rx, const float* __restrict ry,
										const int w, const int h, const int n, const int m)
{
	//Global WarpID
	int wid    = (threadIdx.x + blockDim.x*blockIdx.x) / WARPSIZE;
	//Block level WarpID
	int widB   = threadIdx.x / WARPSIZE;
	//Lane within warp
	int lane   = threadIdx.x % WARPSIZE;
	//Global Warp Stride
	int stride = blockDim.x*gridDim.x / WARPSIZE;

	//Stores per block, per warp data
	static __shared__ float4 shared[32];

	// Process cell one warp at a time
	while(wid < (m-n))
	{
		//First lane checks for valid cell
		if (lane == 0)
		{
			if (!isnan(rx[wid]))
			{
				shared[widB] = make_float4(x[wid+n], y[wid+n], rx[wid], ry[wid]);
				int old = atomicSub(index, 1);
			} else
			{
				shared[widB] = make_float4(HIP_NAN_F, HIP_NAN_F, HIP_NAN_F, HIP_NAN_F);
			}
		}
		__syncthreads();

		//All threads load from shared into registers
		float4 cell = shared[widB];
		if (!isnan(cell.x))
		{
			//Draw cell
			int xC  = (int)cell.x;
			int yC  = (int)cell.y;
			int rxC = (int)cell.z;
			int ryC = (int)cell.w;

			//Horizontal Edge through yC
			for (int ii=xC-rxC + lane; ii < xC+rxC; ii+=WARPSIZE)
				setRedHue(255, dst[yC*w + ii]);

			//Vertical Edge Through xC
			for (int ii=yC-ryC + lane; ii < yC+ryC; ii+=WARPSIZE)
				setRedHue(255, dst[ii*w + xC]);
		}
		wid += stride;
	}

}
// Random Number Generators //
// Generate 2D uniform random values
__global__ void generate_uniform2D_kernel(float* noiseX, float* noiseY, int seed, const int w, const int h, const int n)
{
	int idx        = threadIdx.x + blockIdx.x * blockDim.x;
	int numThreads = blockDim.x*gridDim.x;

	hiprandState localState;

	/* Each thread gets different seed, a different sequence number, no offset */
	hiprand_init(seed, idx, 0, &localState);

	/* Generate pseudo-random normals */
	for(int i = idx; i < n; i+=numThreads)
	{
		// Generate and store
		noiseX[i] = (float)w*hiprand_uniform(&localState);
		noiseY[i] = (float)h*hiprand_uniform(&localState);
	}
}
// Generate 2D uniform random coordinate and filter values
__global__ void generate_uniform2Dfilter_kernel(float* noiseX, float* noiseY, float* score, int seed, const int w, const int h, const int n)
{
	//Scores are random uniform [0 1)
	int idx        = threadIdx.x + blockIdx.x * blockDim.x;
	int numThreads = blockDim.x*gridDim.x;

	hiprandState localState;

	/* Each thread gets different seed, a different sequence number, no offset */
	hiprand_init(seed, idx, 0, &localState);

	/* Generate pseudo-random normals */
	for(int i = idx; i < n; i+=numThreads)
	{
		// Generate and store
		noiseX[i] = (float)w*hiprand_uniform(&localState);
		noiseY[i] = (float)h*hiprand_uniform(&localState);
		score[i]  = hiprand_uniform(&localState);
	}
}

// Quad Tree Routines //
__global__ void reset_arrays_kernel(int* mutex, float* x, float* y, float* rx, float* ry, int* child, int* index, float* left, float* right, float* bottom, float* top, int n, int m)
{
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	int stride = blockDim.x*gridDim.x;
	int offset = 0;

	// reset quadtree arrays
	while(idx + offset < m)
	{  
#pragma unroll 4
		for(int i=0;i<4;i++)
		{
			child[(idx + offset)*4 + i] = -1;
		}
		if(idx + offset >= n)
		{
			x[idx + offset] = HIP_NAN_F;
			y[idx + offset] = HIP_NAN_F;
			rx[idx + offset - n] = HIP_NAN_F;
			ry[idx + offset - n] = HIP_NAN_F;
		}
		offset += stride;
	}

	if(idx == 0)
	{
		*mutex = 0;
		*index = n;
		*left = HIP_INF_F;
		*right = -HIP_INF_F;
		*bottom = HIP_INF_F;
		*top = -HIP_INF_F;
	}
}
__global__ void reset_arrays_kernel(int* mutex, float* x, float* y, float* rx, float* ry, int* child, int* index, float* left, float* right, float* bottom, float* top, const int w, const int h, int n, int m)
{
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	int stride = blockDim.x*gridDim.x;
	int offset = 0;

	// reset quadtree arrays
	while(idx + offset < m)
	{  
#pragma unroll 4
		for(int i=0;i<4;i++)
		{
			child[(idx + offset)*4 + i] = -1;
		}
		if(idx + offset >= n)
		{
			x[idx + offset] = HIP_NAN_F;
			y[idx + offset] = HIP_NAN_F;
			rx[idx + offset - n] = HIP_NAN_F;
			ry[idx + offset - n] = HIP_NAN_F;
		}
		offset += stride;
	}

	//To ensure the write below doesn't get overwritten from above
	__threadfence();

	//Set bounds to image bounds
	if(idx == 0)
	{
		*mutex = 0;
		*index = n+1;	//Set to n + 1 to allow for root
		*left = 0.0f;
		*right = (float)w;
		*bottom = 0.0f;
		*top = (float)h;
		//set root coordinates
		//Create a new cell, starting at index n
		x[n]  = 0.5f*(float)w;
		y[n]  = 0.5f*(float)h;
		rx[0] = 0.5f*(float)w;
		ry[0] = 0.5f*(float)h;
	}
}
 
__global__ void reset_filter_arrays_kernel(int* mutex, float* x, float* y, float* score, float* xf, float* yf, float* scoref,
											float* rx, float* ry, int* child, int* index, float* left, float* right, float* bottom, float* top,
											const int f, int n, int m)
{
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	int stride = blockDim.x*gridDim.x;
	int offset = 0;

	// reset quadtree arrays
	while(idx + offset < m)
	{  
#pragma unroll 4
		for(int i=0;i<4;i++)
		{
			child[(idx + offset)*4 + i] = -1;
		}
		if (idx + offset < f)
		{
			xf[idx + offset] = HIP_NAN_F;
			yf[idx + offset] = HIP_NAN_F;
			scoref[idx + offset]  = HIP_NAN_F;
		}
		if(idx + offset >= n)
		{
			x[idx + offset] = HIP_NAN_F;
			y[idx + offset] = HIP_NAN_F;
			score[idx + offset]  = HIP_NAN_F;
			rx[idx + offset - n] = HIP_NAN_F;
			ry[idx + offset - n] = HIP_NAN_F;
		}
		offset += stride;
	}

	if(idx == 0)
	{
		*mutex = 0;
		*index = n;
		*left = HIP_INF_F;
		*right = -HIP_INF_F;
		*bottom = HIP_INF_F;
		*top = -HIP_INF_F;
	}
}

__global__ void reset_filter_arrays_kernel(int* mutex, float* x, float* y, float* score, float* xf, float* yf, float* scoref,
											float* rx, float* ry, int* child, int* index, float* left, float* right, float* bottom, float* top,
											const int f, const int w, const int h, int n, int m)
{
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	int stride = blockDim.x*gridDim.x;
	int offset = 0;

	// reset quadtree arrays
	while(idx + offset < m)
	{  
#pragma unroll 4
		for(int i=0;i<4;i++)
		{
			child[(idx + offset)*4 + i] = -1;
		}
		if (idx + offset < f)
		{
			xf[idx + offset] = HIP_NAN_F;
			yf[idx + offset] = HIP_NAN_F;
			scoref[idx + offset]  = HIP_NAN_F;
		}
		if(idx + offset >= n)
		{
			x[idx + offset] = HIP_NAN_F;
			y[idx + offset] = HIP_NAN_F;
			rx[idx + offset - n] = HIP_NAN_F;
			ry[idx + offset - n] = HIP_NAN_F;
		}
		offset += stride;
	}

	//To ensure the write below doesn't get overwritten from above
	__threadfence();

	//Set bounds to image bounds
	if(idx == 0)
	{
		*mutex  = 0;
		*index  = n+1;	//Set to n + 1 to allow for root
		*left   = 0.0f;
		*right  = (float)w;
		*bottom = 0.0f;
		*top    = (float)h;
		//set root coordinates
		//Create a new cell, starting at index n
		x[n]  = 0.5f*(float)w;
		y[n]  = 0.5f*(float)h;
		rx[0] = 0.5f*(float)w;
		ry[0] = 0.5f*(float)h;
	}
}

__global__ void compute_bounding_box_kernel(int* mutex, int* index, float* x, float* y, float* rx, float* ry, volatile float* left, volatile float* right, volatile float* bottom, volatile float* top, int n)
{
	//TODO: optimize using warps

	int idx     = threadIdx.x + blockDim.x*blockIdx.x;
	int stride  = blockDim.x*gridDim.x;
	float x_min = x[idx];
	float x_max = x[idx];
	float y_min = y[idx];
	float y_max = y[idx];
	
	__shared__ float left_cache[NTHREADS];
	__shared__ float right_cache[NTHREADS];
	__shared__ float bottom_cache[NTHREADS];
	__shared__ float top_cache[NTHREADS];


	int offset = stride;
	while(idx + offset < n){
		x_min = fminf(x_min, x[idx + offset]);
		x_max = fmaxf(x_max, x[idx + offset]);
		y_min = fminf(y_min, y[idx + offset]);
		y_max = fmaxf(y_max, y[idx + offset]);
		offset += stride;
	}

	left_cache[threadIdx.x] = x_min;
	right_cache[threadIdx.x] = x_max;
	bottom_cache[threadIdx.x] = y_min;
	top_cache[threadIdx.x] = y_max;

	__syncthreads();

	//////////////////////////
	// BLOCK-WISE REDUCTION //
	//////////////////////////

	// NOTE: This could be done by warps

	// assumes blockDim.x is a power of 2!
	int i = blockDim.x/2;
	while(i != 0){
		if(threadIdx.x < i){
			left_cache[threadIdx.x]   = fminf(left_cache[threadIdx.x], left_cache[threadIdx.x + i]);
			right_cache[threadIdx.x]  = fmaxf(right_cache[threadIdx.x], right_cache[threadIdx.x + i]);
			bottom_cache[threadIdx.x] = fminf(bottom_cache[threadIdx.x], bottom_cache[threadIdx.x + i]);
			top_cache[threadIdx.x]    = fmaxf(top_cache[threadIdx.x], top_cache[threadIdx.x + i]);
		}
		__syncthreads();
		i /= 2;
	}

	/////////////////////
	// FINAL REDUCTION //
	/////////////////////

	//NOTE: threadIdx.x == 0 in each block performs final reduction using atomics

	// How the lock works
	// -If a thread has the lock, the mutex will be 1, and the thread loops (spin lock)
	// -If a thread does not have the lock, it takes the lock and is done

	//TODO: Optimize using registers
	if(threadIdx.x == 0){
		while (atomicCAS(mutex, 0 ,1) != 0); // lock
		*left   = fminf(*left, left_cache[0]);
		*right  = fmaxf(*right, right_cache[0]);
		*bottom = fminf(*bottom, bottom_cache[0]);
		*top    = fmaxf(*top, top_cache[0]);
		//set root coordinates
		__threadfence();
		//Create a new cell, starting at idx n
		int cell   = atomicAdd(index,1);
		x[cell]    = 0.5f*(*left + *right);
		y[cell]    = 0.5f*(*top + *bottom);
		rx[cell-n] = 0.5f*(*left - *right);
		ry[cell-n] = 0.5f*(*top - *bottom);
		atomicExch(mutex, 0); // unlock
	}
}


__global__ void build_tree_kernel(volatile float *x, volatile float *y, float* rx, float* ry, volatile int *child, int *index,
									const float *left, const float *right, const float *bottom, const float *top,
									const int n, const int m)
{
	/*
	This routine combines building the Quad Tree with summarizing internal node information
	index:	a global index start at n
	n:		the number of data
	m:		the number of possible nodes
	*/

	int idx    = threadIdx.x + blockIdx.x*blockDim.x;
	int stride = blockDim.x*gridDim.x;

	// build quadtree
	float l;
	float r;
	float b;
	float t;
	int childPath;
	int node;

	bool newBody  = true;
	float posX, posY;
	while(idx < n){

		if(newBody){
			newBody = false;
			//Top/Down Traversal: All particles start in one of the top 4 quads
			l = *left;
			r = *right;
			b = *bottom;
			t = *top;

			node      = n;
			childPath = 0;
			posX      = x[idx];
			posY      = y[idx];

			//Check body location within the top 4 nodes
			if(posX < 0.5*(l+r)){
				childPath += 1;
				r = 0.5*(l+r);
			}
			else{
				l = 0.5*(l+r);
			}
			if(posY < 0.5*(b+t)){
				childPath += 2;
				t = 0.5*(t+b);
			}
			else{
				b = 0.5*(t+b);
			}
		}

		//Set childIndex, which could be after mutliple loops
		int childIndex = child[node*4 + childPath];

		// traverse tree until we hit leaf node (could be allocated or not)

		//NOTE: childIndex >= n means we are in a cell not a leaf
		// You could also land in an unallocated (-1) or locked (-2) node
		while(childIndex >= n){
			//Check body location within the 4 quads of this node
			node = childIndex;
			childPath = 0;
			if(posX < 0.5*(l+r)){
				childPath += 1;
				r = 0.5*(l+r);
			}
			else{
				l = 0.5*(l+r);
			}
			if(posY < 0.5*(b+t)){
				childPath += 2;
				t = 0.5*(t+b);
			}
			else{
				b = 0.5*(t+b);
			}

			//Advance to child of this cell
			childIndex = child[4*node + childPath];
		}

		//At this point childIndex: [-1 n]

		// Check if child is already locked i.e. childIndex == -2
		if(childIndex != -2){
			//Acquire lock, which is only possible if child[locked]: [-1 n]
			int locked = node*4 + childPath;
			if(atomicCAS((int*)&child[locked], childIndex, -2) == childIndex){
				//If unallocated, insert body and unlock
				if(childIndex == -1){
					// Insert body and release lock
					child[locked] = idx;
				}
				else{
					//Sets max on number of cells
					int patch = 4*n;
					while(childIndex >= 0){

						// childIndex should always be -1, unallocated, or >=0, allocated

						//Create a new cell, starting at index n
						int cell = atomicAdd(index,1);

						//Compare against maximum allowable cells
						patch = min(patch, cell);

						//If the maximum number of cells have been reached:
						// It prunes away the node above
						if(patch != cell){
							child[4*node + childPath] = cell;
						}

						// insert old particle into new cell
						childPath = 0;
						if(x[childIndex] < 0.5*(l+r)){
							childPath += 1;
						}
						if(y[childIndex] < 0.5*(b+t)){
							childPath += 2;
						}

						#ifdef DEBUG
							// if(cell >= 2*n){
							if(cell >= m){
								printf("%s\n", "error cell index is too large!!");
								printf("cell: %d\n", cell);
							}
						#endif

						//Assign old particle to subtree leaf
						child[4*cell + childPath] = childIndex;

						//SET ROOT OF NEW CELL AND LENGTH OF SIDES
						x[cell]    = 0.5*(l+r);
						y[cell]    = 0.5*(b+t);
						rx[cell-n] = 0.5*(r-l);
						ry[cell-n] = 0.5*(t-b);

						// insert new particle
						node = cell;
						childPath = 0;
						if(posX < 0.5*(l+r)){
							childPath += 1;
							r = 0.5*(l+r);
						}
						else{
							l = 0.5*(l+r);
						}
						if(posY < 0.5*(b+t)){
							childPath += 2;
							t = 0.5*(t+b);
						}
						else{
							b = 0.5*(t+b);
						}

						//Set to value of child at this entry, which could be:
						// -1 == break
						// > n if new data landed in the same part of the sub-tree as old data
						childIndex = child[4*node + childPath];
					}

					//This means childIndex is set to -1, unallocated, so allocated as body Index
					child[4*node + childPath] = idx;

					__threadfence();  // Ensures all writes to global memory are complete before lock is released

					//Release lock and replace leaf with this cell
					child[locked] = patch;
				}	// if(childIndex == -1): first assignment to body or not

				//Advance to next body
				idx += stride;
				newBody    = true;
			}	//if(atomicCAS((int*)&child[locked], childIndex, -2) == childIndex)

		}	//if(childIndex != -2): locked already or not. If locked, go around again

		// Wait for threads in block to release locks to reduce memory pressure
		__syncthreads(); // not needed for correctness
	}
}

__global__ void filter_tree_kernel(volatile float* x, volatile float* y, volatile float* score,
									float* rx, float* ry, volatile int* child, int* index,
									const float* left, const float* right, const float* bottom, const float* top,
									const int n, const int m, const int d, const int f)
{
	/*
	This routine combines building the Quad Tree with spatial filtering and summarizing internal node information
	index:	a global index start at n
	n:		the number of possible data
	m:		the number of possible nodes
	d:		the number of current data
	f:		the maximum number of cells to be created, which limits data by single occupancy filter
	*/

	int idx    = threadIdx.x + blockIdx.x*blockDim.x;
	int stride = blockDim.x*gridDim.x;

	// build quadtree
	float l;
	float r;
	float b;
	float t;
	int childPath;
	int node;

	bool newBody  = true;
	float posX, posY;
	while(idx < d){

		if(newBody){
			newBody = false;
			//Top/Down Traversal: All particles start in one of the top 4 quads
			l = *left;
			r = *right;
			b = *bottom;
			t = *top;

			node      = n;
			childPath = 0;
			posX      = x[idx];
			posY      = y[idx];

			//Check body location within the top 4 nodes
			if(posX < 0.5*(l+r)){
				childPath += 1;
				r = 0.5*(l+r);
			}
			else{
				l = 0.5*(l+r);
			}
			if(posY < 0.5*(b+t)){
				childPath += 2;
				t = 0.5*(t+b);
			}
			else{
				b = 0.5*(t+b);
			}
		}

		//Set childIndex, which could be after mutliple loops
		int childIndex = child[node*4 + childPath];

		// traverse tree until we hit leaf node (could be allocated or not)

		//NOTE: childIndex >= d means we are in a cell not a leaf
		// You could also land in an unallocated (-1) or locked (-2) node
		while(childIndex >= d){
			//Check body location within the 4 quads of this node
			node = childIndex;
			childPath = 0;
			if(posX < 0.5*(l+r)){
				childPath += 1;
				r = 0.5*(l+r);
			}
			else{
				l = 0.5*(l+r);
			}
			if(posY < 0.5*(b+t)){
				childPath += 2;
				t = 0.5*(t+b);
			}
			else{
				b = 0.5*(t+b);
			}

			//Advance to child of this cell
			childIndex = child[4*node + childPath];
		}

		printf("Thread %d at node %d is %d\n", idx, 4*node + childPath, childIndex);
		//At this point childIndex: [-1 d]

		// Check if child is already locked i.e. childIndex == -2
		if(childIndex != -2){
			//Acquire lock, which is only possible if child[locked]: [-1 d]
			int locked = 4*node + childPath;
			if(atomicCAS((int*)&child[locked], childIndex, -2) == childIndex){
				//If unallocated, insert body and unlock
				if(childIndex == -1){
					// Insert body and release lock
					child[locked] = idx;
					printf("Initializing with %02d at [%f, %f]\n", idx, x[idx], y[idx]);
				}
				else{
					//Sets max on number of cells
					int patch = 4*n;
					bool bMoreCells = true;

					//for handling the case of new and old data landing in same node
					int parentCell  = -1;
					int tmpIdx;
					while(childIndex >= 0)
					{
						// childIndex should always be -1, unallocated, or >=0, allocated

						//Create a new cell, starting at index n
						int cell = atomicAdd(index,1);

						//Compare against maximum allowable cells
						patch = min(patch, cell);

						//If f cells already created, filter by response
						if (cell - n >= f)
						{
							// printf("Cell is %d and f is %d\n", cell -n, f);
							int keeper = idx;
							if (score[childIndex] < score[idx])
							{
								// Replace data and release lock
								printf("\tSwapping %d with %d\n", childIndex, idx);
								keeper = idx;
							} else
							{
								//... or put it back to unlock
								printf("\tKeeping %d over %d\n", childIndex, idx);
								keeper = childIndex;
							}
							//Check for the case of new and old data landing in same node
							if (parentCell > 0)
							{
								child[tmpIdx] = keeper;
								__threadfence();
								child[locked] = parentCell;
							} else
							{
								child[locked] = keeper;
							}

							bMoreCells = false;
							break;
						}

						//If the maximum number of cells have been reached:
						// It prunes away the node above
						if(patch != cell){
							child[4*node + childPath] = cell;
						}

						// insert old particle into new cell
						childPath = 0;
						if(x[childIndex] < 0.5*(l+r)){
							childPath += 1;
						}
						if(y[childIndex] < 0.5*(b+t)){
							childPath += 2;
						}

						#ifdef DEBUG
							// if(cell >= 2*n){
							if(cell >= m){
								printf("%s\n", "error cell index is too large!!");
								printf("cell: %d\n", cell);
							}
						#endif

						//Assign old particle to subtree leaf
						child[4*cell + childPath] = childIndex;

						//SET ROOT OF NEW CELL AND LENGTH OF SIDES
						x[cell]    = 0.5*(l+r);
						y[cell]    = 0.5*(b+t);
						rx[cell-n] = 0.5*(r-l);
						ry[cell-n] = 0.5*(t-b);

						// insert new particle
						parentCell = cell;
						node       = cell;
						childPath = 0;
						if(posX < 0.5*(l+r)){
							childPath += 1;
							r = 0.5*(l+r);
						}
						else{
							l = 0.5*(l+r);
						}
						if(posY < 0.5*(b+t)){
							childPath += 2;
							t = 0.5*(t+b);
						}
						else{
							b = 0.5*(t+b);
						}

						//Set to value of child at this entry, which could be:
						// -1 == break
						// > n if new data landed in the same part of the sub-tree as old data
						tmpIdx     = 4*node + childPath;
						childIndex = child[tmpIdx];
					}

					if (bMoreCells)
					{
						//This means childIndex is set to -1, unallocated, so allocated as body Index
						printf("Initializing NEW with %02d at [%f, %f]\n", idx, x[idx], y[idx]);
						child[4*node + childPath] = idx;

						__threadfence();  // Ensures all writes to global memory are complete before lock is released

						//Release lock and replace leaf with this cell
						printf("Releasing lock as %d\n", patch);
						child[locked] = patch;
					}
				}	// if(childIndex == -1): first assignment to body or not

				//Advance to next body
				idx += stride;
				newBody    = true;
			}	//if(atomicCAS((int*)&child[locked], childIndex, -2) == childIndex)

		}	//if(childIndex != -2): locked already or not. If locked, go around again

		// Wait for threads in block to release locks to reduce memory pressure
		__syncthreads(); // not needed for correctness
	}

	__syncthreads();
	if (threadIdx.x + blockIdx.x*blockDim.x == 0)
	{
		for (int i = 0; i < 16; i++){
			printf("%d, ", child[4*n+i]);
		}
		printf("\n");
	}
}

__global__ void pack_filtered_data_kernel(float* xf, float* yf, float* scoref,
											float* x, float* y, float* score,
											int* child, const int n, const int d, const int q)
{
	/*
	Data filtered through the Quad Tree are scattered across the child array. We need to pack
	them into xf, yf, and scoref.

	It uses depth-first search.

	Marking leaves or internal cells as -1 means they have been fully processed (unallocated again)

	index:	a global index start at n
	n:		root of tree node index
	d:		the number of current data
	q:		the number of filtered data
	*/

	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if (idx >= q)
		return;

	int parentIndex;
	int parentNode;

	//Every thread will hit a leaf
	int  childIndex;
	bool notAtTop = false;

	int iterations = 0;
	while(true)
	{
		iterations++;

		//Start at the top of the tree
		if (!notAtTop)
		{
			//Start at Root Node
			parentNode = n;

			//Inspect children of root for initial parentIndex
			// - At least one of them should always be active
			for (int i = 0; i< 4; i++)
			{
				//Returns the NEXT parent node when indexing child array
				parentIndex = 4*parentNode + i;

				childIndex  = child[parentIndex];
				if (childIndex > 0)
				{
					//Advance down the tree and assign new parent node
					parentNode = childIndex;
					break;
				}
			}
		}

		//Leaf check
		if ((childIndex < d) && (childIndex >=0))
		{
			//We're at a leaf, so set to unallocated = -1
			if(atomicCAS((int*)&child[parentIndex], childIndex, -1) == childIndex)
			{
				//This thread is the first here, so the childIndex goes with it
				break;
			} else
			{
				//This thread didn't get here fast enough, so it has to start over
				notAtTop = false;
				continue;
			}
		}

		//Inspect children of this parent cell
		notAtTop = false;	//assume all children are done
		for (int i = 0; i< 4; i++)
		{
			int tmpIdx = 4*parentNode + 1;
			childIndex = child[tmpIdx];
			if (childIndex > 0)
			{
				//Advance to next level down tree
				notAtTop    = true;
				parentIndex = tmpIdx;
				parentNode  = childIndex;
				break;
			}
		}

		//If all children are done, then mark parent as done and go back to the top
		if (!notAtTop)
		{
			//It doesn't matter which thread gets here first
			atomicExch((int*)&child[parentIndex], -1);
		}

		//DEBUGGING
		#ifdef DEBUG
		if (iterations > 5*q)
		{
			printf("Thread %d has gone around %d times. Breaking...\n", idx, iterations);
			//dummy value to ensure no segfault
			childIndex = 0;
			break;
		}
		#endif
	}

	printf("thread %d writing out childIndex %d\n", idx, childIndex);

	//Write out into packed array
	xf[idx]     = x[childIndex];
	yf[idx]     = y[childIndex];
	scoref[idx] = score[childIndex];
}

} // namespace quadTreeKernels
