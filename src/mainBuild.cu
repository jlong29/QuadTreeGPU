#include "hip/hip_runtime.h"
//STL
#include <stdio.h>
#include <iostream>
#include <string.h>
#include <time.h>

//CUDA
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_functions.h>    // sdkCreateTimer: contains definitions in header

//OPENGL
#include <helper_gl.h>
#include <GL/freeglut.h>

//CUDA OpenGL interoperability
#include <cuda_gl_interop.h>
#define REFRESH_DELAY     10 //ms

#include "QuadTreeBuilder.h"

//Reset Quad Tree state with new data
static bool bReset = false;

// GLOBAL Variables //
//Problem Size
//Parameters
int N;
int W;
int H;

int window_width;
int window_height;
float aspRat;

//The star of the show
QuadTreeBuilder quadTree;

//OPENGL
//Texture variables
GLuint imageTex;
struct hipGraphicsResource *pcuImageRes;

// Timing Code
//OpenGL loop = GPU + host
StopWatchInterface *timer;
int fpsCount;        // FPS count for averaging
int fpsLimit;        // FPS limit for sampling
float avgFPS;
uint frameCount;

/////////////////////////////////
// OPENGL FORWARD DECLARATIONS //
/////////////////////////////////
bool initGL(int *argc, char **argv);

//IMAGE DATA
void display();
void keyboard(unsigned char key, int x, int y);
void cleanup();
void timerEvent(int value);

//Help Information
static void show_usage(std::string name)
{
	std::cerr << "Usage: " << name << " <options(s)>"
			  << "Options:\n"
			  << "\t-i,--help\tShow this help message\n"
			  << "\t-n,\t\tset the number of data points to generate\n"
			  << "\t-w,\t\tset the width of the image plane\n"
			  << "\t-h,\t\tset the height of the image plane\n"
			  << std::endl;
}

int runBuild();

int main(int argc, char** argv)
{
	//Input Parameters
	N = 16;
	W = 640;
	H = 480;

	for (int i = 1; i < argc; ++i)
	{
		std::string arg = argv[i];
		if ((arg == "-i") || (arg == "--help"))
		{
			show_usage(argv[0]);
			return 0;  
		} else if (arg == "-n")
		{
			if (i + 1 < argc)
			{
				N = (size_t)atoi(argv[++i]);
			} else
			{
				fprintf(stderr, "-n option requires one argument indicating a sample size.\n");
				return -1;
			}
		} else if (arg == "-w")
		{
			if (i + 1 < argc)
			{
				W = (size_t)atoi(argv[++i]);
			} else
			{
				fprintf(stderr, "-w option requires one argument indicating an image width.\n");
				return -1;
			}
		} else if (arg == "-h")
		{
			if (i + 1 < argc)
			{
				H = (size_t)atoi(argv[++i]);
			} else
			{
				fprintf(stderr, "-h option requires one argument indicating an image height\n");
				return -1;
			}
		}
	}

	window_width  = W;
	window_height = H;

	//Set QuadTreeBuilder parameters
	quadTree.setParameters(N, W, H);

	//initialize timers
	fpsCount   = 0;
	fpsLimit   = 1;
	avgFPS     = 0.0f;
	frameCount = 0;

	/* Set Up */
	//Set Device
	int deviceCount;
	checkCudaErrors(hipGetDeviceCount(&deviceCount));
	if (deviceCount == 0) {
		printf("There is no device supporting CUDA\n");
		return EXIT_FAILURE;
	}

	int dev = 0;
	if (dev >= deviceCount){
		printf("Input error: dev >= deviceCount\n");
		return EXIT_FAILURE;
	}

	hipSetDevice(dev);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);

	printf("\tDevice %d: \"%s\"\n", dev, deviceProp.name);

	// OpenGL: initialize on this device and set up windows
	if (false == initGL(&argc, argv))
	{
		return -1;
	}

	// Create the CUTIL timer
	sdkCreateTimer(&timer);

	//Allocate Memory
	if (quadTree.allocate() < 0)
	{
		cleanup();
		return -1;
	}

	if (runBuild() < 0)
	{
		cleanup();
		return -1;
	}

	//convert device memory to texture
	hipArray_t ArrIm;
	hipGraphicsMapResources(1, &pcuImageRes, 0);
	hipGraphicsSubResourceGetMappedArray(&ArrIm, pcuImageRes, 0, 0);

	checkCudaErrors(hipMemcpyToArray(ArrIm, 0, 0, quadTree.d_img, quadTree.imgSz, hipMemcpyDeviceToDevice));
	hipGraphicsUnmapResources(1, &pcuImageRes, 0);

	////////////////////////////////
	// LAUNCH OPENGL DISPLAY LOOP //
	////////////////////////////////
	glutMainLoop();
	return 0;
}

//OpenGL function definitions
void computeFPS()
{
	frameCount++;
	fpsCount++;

	if (fpsCount == fpsLimit)
	{
		avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
		fpsCount = 0;
		fpsLimit = (int)MAX(avgFPS, 1.f);

		sdkResetTimer(&timer);
	}
	
	char fps[256];
	sprintf(fps, "Quad Tree: %3.1f fps", avgFPS);
	glutSetWindowTitle(fps);
}

bool initGL(int *argc, char **argv)
{
	glutInit(argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);

	//Create Image Data Window
	glutInitWindowSize(window_width, window_height);
	glutCreateWindow("Quad Tree");
	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	glutReportErrors();

	// NOTE: These calls don't work until first window is created

	//initialize necessary OpenGL extensions
	if (!isGLVersionSupported(2,0))
	{
		fprintf(stderr,"ERROR: Support for necessary OpenGL extensions missing.");
		fflush(stderr);
		return false;
	}
	fprintf(stdout,"OpenGL version supported by this platform: (%s)\n", glGetString(GL_VERSION));

	// Create and Register OpenGL Texture for Image(1 channel)
	hipError_t err1;
	glGenTextures(1, &imageTex);
	glBindTexture(GL_TEXTURE_2D, imageTex);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, window_width, window_height, 0, GL_BGRA, GL_UNSIGNED_BYTE, NULL);
	glBindTexture(GL_TEXTURE_2D, 0);

	err1 = hipGraphicsGLRegisterImage(&pcuImageRes, imageTex, GL_TEXTURE_2D, cudaGraphicsMapFlagsWriteDiscard);

	if (err1 != 0)
	{
		fprintf(stderr,"ERROR: Registering openGL texture failed\n");
		return false;
	}
	glutReportErrors();

	// Image Data: register callbacks
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutTimerFunc(REFRESH_DELAY, timerEvent, 0);
	glutCloseFunc(cleanup);

	return true;
}

void display()
{
	//Time it
	sdkStartTimer(&timer);

	if (bReset)
	{
		bReset = false;

		if (runBuild() < 0)
		{
			cleanup();
			return;
		}

		//convert device memory to texture
		hipArray_t ArrIm;
		hipGraphicsMapResources(1, &pcuImageRes, 0);
		hipGraphicsSubResourceGetMappedArray(&ArrIm, pcuImageRes, 0, 0);

		//NOTE: DISTORTION MODEL: 3 * d_dstReSz
		checkCudaErrors(hipMemcpyToArray(ArrIm, 0, 0, quadTree.d_img, quadTree.imgSz, hipMemcpyDeviceToDevice));
		hipGraphicsUnmapResources(1, &pcuImageRes, 0);
	}

	/////////////////////////
	// DISPLAY WITH OPENGL //
	/////////////////////////

	//OpenGL Part
	glClear(GL_COLOR_BUFFER_BIT);
	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);

	glBindTexture(GL_TEXTURE_2D, imageTex);
	glEnable(GL_TEXTURE_2D);
	glDisable(GL_DEPTH_TEST);
	glDisable(GL_LIGHTING);
	glTexEnvf(GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_REPLACE);

	//Note: texCoords go clockwise from top left and verts go counter-clockwise from lower left
	glBegin(GL_QUADS);
	glTexCoord2f(0.0, 1.0);
	glVertex3f(-1, -1.0, 0.0);
	glTexCoord2f(1.0, 1.0);
	glVertex3f(1, -1.0, 0.0);
	glTexCoord2f(1.0, 0.0);
	glVertex3f(1, 1.0, 0.0);
	glTexCoord2f(0.0, 0.0);
	glVertex3f(-1, 1.0, 0.0);
	glEnd();
	glBindTexture(GL_TEXTURE_2D, 0);
	glDisable(GL_TEXTURE_2D);

	glutReportErrors();

	hipDeviceSynchronize();

	// Updating timing information
	sdkStopTimer(&timer);
	computeFPS();

	//swap
	glutSwapBuffers();
}

//Keyboard callback
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
	fprintf(stdout,"\tKey Press: %u\n", (uint)key);
	switch (key)
	{
	case (27) :
	{
		glutDestroyWindow(glutGetWindow());
		return;
	}
	case 'r':
	{
		//r: Reset Quad Tree with new data
		bReset = true;
		return;
	}
	}
}

void timerEvent(int value)
{
	if (glutGetWindow())
	{
		glutPostRedisplay();
		glutTimerFunc(REFRESH_DELAY, timerEvent,0);
	}
}

void cleanup()
{
	sdkDeleteTimer(&timer);

	// CUDA/OPENGL
	fprintf(stdout,"\tCUDA:\n");
	hipGraphicsUnregisterResource(pcuImageRes);
	glDeleteTextures(1, &imageTex);
	imageTex = 0;
	fprintf(stdout,"\t\tAll openGL resources cleaned\n");

	//Deallocate device memory and destory timers
	quadTree.deallocate();

	fprintf(stdout,"\t\tAll Cuda resources cleaned\n");
}

int runBuild()
{
	//Set random data
	if (quadTree.resetData() < 0)
	{
		return -1;
	}

	// BUILD QUAD TREE
	quadTree.build();

	//Visualize output
	if (quadTree.createBuildViz() < 0)
	{
		return -1;
	}

	//Download data
	if (quadTree.downloadData() < 0)
	{
		return -1;
	}

	printf("Printing out 2D Noise:\n\t");
	for (int ii = 0; ii < min(100, N); ii++)
	{
		printf("[%d, %d], ", (int)quadTree.h_x[ii], (int)quadTree.h_y[ii]);
	}
	printf("\n");

	return 0;
}
