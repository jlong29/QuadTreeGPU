#include "hip/hip_runtime.h"
//StL
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <time.h>
#include <math.h>

//CUDA
#include "QuadTreeBuilder.h"
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

#include "quadTreeKernels.h"

namespace quadTreeGPU
{

// #define TIMINGDEBUG

using namespace quadTreeKernels;

QuadTreeBuilder::QuadTreeBuilder()
{
	width    = -1;
	height   = -1;
	numData  = -1;

	numTestData      = -1;
	numFilteredData  = -1;
	
	d_left   = NULL;
	d_right  = NULL;
	d_bottom = NULL;
	d_top    = NULL;

	d_x      = NULL;
	d_y      = NULL;
	d_score  = NULL;
	d_rx     = NULL;
	d_ry     = NULL;

	d_xf     = NULL;
	d_yf     = NULL;
	d_scoref = NULL;

	d_child  = NULL;

	d_index  = NULL;
	d_mutex  = NULL;

	h_x      = NULL;
	h_y      = NULL;

	h_xf	 = NULL;
	h_yf     = NULL;

	d_img    = NULL;

	timersCreated = false;
}

QuadTreeBuilder::QuadTreeBuilder(int n, int w, int h, int q, int d):
	numData(n),
	width(w),
	height(h),
	numFilteredData(q),
	numTestData(d)
{
	numNodes   = 2*n+12000;	// A magic large function of n
	cellMargin = 2.0f;

	// allocate host data
	dataSz   = numData*sizeof(float);
	nodeSz   = numNodes*sizeof(float);
	imgSz    = width*height*sizeof(uchar4);

	//GPU Launch Configurations
	//1D
	threads = NTHREADS;
	blocks  = divUp(numData, NTHREADS);

	//2D
	blockDim.x = WARPSIZE;
	blockDim.y = NWARPS;

	gridDim.x  = divUp(width, blockDim.x);
	gridDim.y  = divUp(height, blockDim.y);

	// allocate device data
	checkCudaErrors(hipMalloc((void**)&d_left, sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_right, sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_bottom, sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_top, sizeof(float)));
	checkCudaErrors(hipMemset(d_left, 0, sizeof(float)));
	checkCudaErrors(hipMemset(d_right, 0, sizeof(float)));
	checkCudaErrors(hipMemset(d_bottom, 0, sizeof(float)));
	checkCudaErrors(hipMemset(d_top, 0, sizeof(float)));

	checkCudaErrors(hipMalloc((void**)&d_x, nodeSz));
	checkCudaErrors(hipMalloc((void**)&d_y, nodeSz));
	checkCudaErrors(hipMalloc((void**)&d_score, nodeSz));
	checkCudaErrors(hipMalloc((void**)&d_rx, nodeSz-dataSz));	//NOTE; -dataSz
	checkCudaErrors(hipMalloc((void**)&d_ry, nodeSz-dataSz));
	
	checkCudaErrors(hipMalloc((void**)&d_child, 4*numNodes*sizeof(int)));

	checkCudaErrors(hipMalloc((void**)&d_index, sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&d_mutex, sizeof(int)));
	
	h_x      = new float[numNodes];
	h_y      = new float[numNodes];
	checkCudaErrors(hipMalloc((void**)&d_img, imgSz));

	if (numFilteredData > 0)
	{	
		checkCudaErrors(hipMalloc((void**)&d_xf, numFilteredData*sizeof(float)));
		checkCudaErrors(hipMalloc((void**)&d_yf, numFilteredData*sizeof(float)));
		checkCudaErrors(hipMalloc((void**)&d_scoref, numFilteredData*sizeof(float)));

		h_xf = new float[numFilteredData];
		h_yf = new float[numFilteredData];
	}

	//Create Timers
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	timersCreated = true;
}

QuadTreeBuilder::~QuadTreeBuilder()
{
	deallocate();
}

int QuadTreeBuilder::allocate()
{
	if (numData < 0)
	{
		fprintf(stderr, "QuadTreeBuilder::allocate(): numData < 0, must initialize prior to allocation\n");
		return -1;
	}

	dataSz   = numData*sizeof(float);

	numNodes = 2*numData+12000;	// A magic large function of n
	nodeSz   = numNodes*sizeof(float);

	cellMargin = 2.0f;

	//GPU Launch Configurations
	//1D
	threads = NTHREADS;
	blocks  = divUp(numData, NTHREADS);

	//2D
	blockDim.x = WARPSIZE;
	blockDim.y = NWARPS;

	gridDim.x  = divUp(width, blockDim.x);
	gridDim.y  = divUp(height, blockDim.y);

	// allocate device data
	if (d_left==NULL)
	{
		checkCudaErrors(hipMalloc((void**)&d_left, sizeof(float)));
		checkCudaErrors(hipMemset(d_left, 0, sizeof(float)));
	}
	if (d_right==NULL)
	{
		checkCudaErrors(hipMalloc((void**)&d_right, sizeof(float)));
		checkCudaErrors(hipMemset(d_right, 0, sizeof(float)));
	}
	if (d_bottom==NULL)
	{
		checkCudaErrors(hipMalloc((void**)&d_bottom, sizeof(float)));
		checkCudaErrors(hipMemset(d_bottom, 0, sizeof(float)));
	}
	if (d_top==NULL)
	{
		checkCudaErrors(hipMalloc((void**)&d_top, sizeof(float)));
		checkCudaErrors(hipMemset(d_top, 0, sizeof(float)));
	}
	if (d_x==NULL)
	{
		checkCudaErrors(hipMalloc((void**)&d_x, nodeSz));
	}
	if (d_y==NULL)
	{
		checkCudaErrors(hipMalloc((void**)&d_y, nodeSz));
	}
	if (d_score==NULL)
	{
		checkCudaErrors(hipMalloc((void**)&d_score, nodeSz));
	}
	if (d_rx==NULL)
	{
		checkCudaErrors(hipMalloc((void**)&d_rx, nodeSz-dataSz));	//NOTE: -dataSz
	}
	if (d_ry==NULL)
	{
		checkCudaErrors(hipMalloc((void**)&d_ry, nodeSz-dataSz));
	}
	if (d_child==NULL)
	{
		checkCudaErrors(hipMalloc((void**)&d_child, 4*numNodes*sizeof(int)));
	}
	if (d_index==NULL)
	{
		checkCudaErrors(hipMalloc((void**)&d_index, sizeof(int)));
	}
	if (d_mutex==NULL)
	{
		checkCudaErrors(hipMalloc((void**)&d_mutex, sizeof(int)));
	}
	if (h_x==NULL)
		h_x      = new float[numNodes];
	if (h_y==NULL)
		h_y      = new float[numNodes];
	if (d_img==NULL)
	{
		imgSz    = width*height*sizeof(uchar4);
		checkCudaErrors(hipMalloc((void**)&d_img, imgSz));
	}

	if (numFilteredData > 0)
	{
		if (d_xf==NULL)
		{
			checkCudaErrors(hipMalloc((void**)&d_xf, numFilteredData*sizeof(float)));
		}
		if (d_yf==NULL)
		{
			checkCudaErrors(hipMalloc((void**)&d_yf, numFilteredData*sizeof(float)));
		}
		if (d_scoref==NULL)
		{
			checkCudaErrors(hipMalloc((void**)&d_scoref, numFilteredData*sizeof(float)));
		}
		if (h_xf==NULL)
			h_xf = new float[numFilteredData];
		if (h_yf==NULL)
			h_yf = new float[numFilteredData];
	}

	//Create Timers
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	timersCreated = true;

	return 0;
}
void QuadTreeBuilder::deallocate()
{
	if (d_left!=NULL)
	{
		checkCudaErrors(hipFree(d_left));
		d_left = NULL;
	}
	if (d_right!=NULL)
	{
		checkCudaErrors(hipFree(d_right));
		d_right = NULL;
	}
	if (d_bottom!=NULL)
	{
		checkCudaErrors(hipFree(d_bottom));
		d_bottom = NULL;
	}
	if (d_top!=NULL)
	{
		checkCudaErrors(hipFree(d_top));
		d_top = NULL;
	}

	if (d_x!=NULL)
	{
		checkCudaErrors(hipFree(d_x));
		d_x = NULL;
	}
	if (d_y!=NULL)
	{
		checkCudaErrors(hipFree(d_y));
		d_y = NULL;
	}
	if (d_score!=NULL)
	{
		checkCudaErrors(hipFree(d_score));
		d_score = NULL;
	}
	if (d_rx!=NULL)
	{
		checkCudaErrors(hipFree(d_rx));
		d_rx = NULL;
	}
	if (d_ry!=NULL)
	{
		checkCudaErrors(hipFree(d_ry));
		d_ry = NULL;
	}
	if (d_child!=NULL)
	{
		checkCudaErrors(hipFree(d_child));
		d_child = NULL;
	}

	if (d_index!=NULL)
	{
		checkCudaErrors(hipFree(d_index));
		d_index = NULL;	
	}
	if (d_mutex!=NULL)
	{
		checkCudaErrors(hipFree(d_mutex));
		d_mutex = NULL;
	}
	if (h_x!=NULL)
	{
		delete [] h_x;
		h_x = NULL;
	}
	if (h_y!=NULL)
	{
		delete [] h_y;
		h_y = NULL;
	}
	if (d_img!=NULL)
	{
		checkCudaErrors(hipFree(d_img));
		d_img = NULL;
	}
	if (d_xf!=NULL)
	{
		checkCudaErrors(hipFree(d_xf));
		d_xf = NULL;
	}
	if (d_yf!=NULL)
	{
		checkCudaErrors(hipFree(d_yf));
		d_yf = NULL;
	}
	if (d_scoref!=NULL)
	{
		checkCudaErrors(hipFree(d_scoref));
		d_scoref = NULL;
	}
	if (h_xf!=NULL)
	{
		delete [] h_xf;
		h_xf = NULL;
	}
	if (h_yf!=NULL)
	{
		delete [] h_yf;
		h_yf = NULL;
	}

	if (timersCreated)
	{
		checkCudaErrors(hipEventDestroy(start));
		checkCudaErrors(hipEventDestroy(stop));
		timersCreated = false;
	}
}

//Set parameters: n is required for functioning. w and h are for visualization
void QuadTreeBuilder::setParameters(int n, int w, int h, int q, int d)
{
	//Set object
	width           = w;
	height          = h;
	numData         = n;
	numTestData     = d;
	numFilteredData = q;
}

//Set data generated by another GPU process
void QuadTreeBuilder::setData(const float* x, const float* y)
{
	checkCudaErrors(hipMemcpy(d_x, x, dataSz, hipMemcpyDeviceToDevice));
	checkCudaErrors(hipMemcpy(d_y, y, dataSz, hipMemcpyDeviceToDevice));
}
void QuadTreeBuilder::setData(const float* x, const float* y, const float* score, const int d)
{
	checkCudaErrors(hipMemcpy(d_x, x, d*sizeof(float), hipMemcpyDeviceToDevice));
	checkCudaErrors(hipMemcpy(d_y, y, d*sizeof(float), hipMemcpyDeviceToDevice));
	checkCudaErrors(hipMemcpy(d_score, score, d*sizeof(float), hipMemcpyDeviceToDevice));
}
void QuadTreeBuilder::setData(float* x, float* y, float* score, const unsigned int* d)
{
	d_setData<<<blocks, threads>>>(d_x, d_y, d_score, x, y, score, d);
}

void QuadTreeBuilder::setCellMargin(const float cm)
{
	cellMargin = cm;
}

//build the quad tree
int QuadTreeBuilder::build()
{
	if (numData < 0)
	{
		fprintf(stderr, "QuadTreeBuilder::build(): numData < 0, must initialize prior to building\n");
		return -1;
	}

	#ifdef TIMINGDEBUG
	checkCudaErrors(hipEventRecord(start,0));
	#endif

	ResetArrays(width, height);
	BuildQuadTree();

	#ifdef TIMINGDEBUG
	checkCudaErrors(hipEventRecord(stop,0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elpsTime, start, stop));
	printf("\n\nElapsed time for QuadTree Build:  %9.6f ms \n", elpsTime);
	#endif

	return 0;
}

//filter the data from d > f points to f points according to highest score
//Takes in host side counter
int QuadTreeBuilder::filter(float* x, float* y, float* score, const int d, const int q)
{
	if (numData < 0)
	{
		fprintf(stderr, "QuadTreeBuilder::filter(): numData < 0, must initialize prior to filtering\n");
		return -1;
	}
	if ((d > numData) || (d <= q))
	{
		fprintf(stderr, "QuadTreeBuilder::filter(): d must be <= numData and > q\n");
		return -1;
	}

	//This a tight upperbound upon cells for a target set size of filtered data
	int f = (int)ceil(((float)q*cellMargin - 1.0f)/3.0f);

	ResetFilterArrays(q, width, height);
	FilterQuadTree(d, q, f);

	return 0;
}
//Operates upon internal state
int QuadTreeBuilder::filter()
{
	return filter(d_x, d_y, d_score, numTestData, numFilteredData);
}

//Takes in device side counter
int QuadTreeBuilder::filter(float* x, float* y, float* score, unsigned int* d, const int q)
{
	if (numData < 0)
	{
		fprintf(stderr, "QuadTreeBuilder::filter(): numData < 0, must initialize prior to filtering\n");
		return -1;
	}

	//This a tight upperbound upon cells for a target set size of filtered data
	int f = (int)ceil(((float)q*cellMargin - 1.0f)/3.0f);

	setData(x, y, score, d);
	ResetFilterArrays(q, width, height);
	FilterQuadTreeDev(d, q, f);

	return 0;
}

//Operates upon internal state and external device data intput
int QuadTreeBuilder::filter(unsigned int* d)
{
	return filter(d_x, d_y, d_score, d, numFilteredData);
}
//Write visualization
int QuadTreeBuilder::createBuildViz()
{
	if ((width<0) || (height<0))
	{
		fprintf(stderr, "QuadTreeBuilder::createBuildViz(): width or height < 0, must initialize prior to vizualization\n");
		return -1;
	}

	//Write Random data onto image buffer
	#ifdef TIMINGDEBUG
	checkCudaErrors(hipEventRecord(start, 0));
	#endif

	d_setBlackImag<<<gridDim, blockDim>>>(d_img, width, height);

	int blocksD = divUp(numNodes - numData, threads);
	std::cout << "BlocksD is " << blocksD << std::endl;
	d_drawCellInnerEdges<<<blocksD, threads>>>(d_img, d_index, d_x, d_y, d_rx, d_ry, width, height, numData, numNodes);

	//Write point last to avoid occulsion by lines (no alpha blending)
	d_writeData2Image<<<blocks, threads>>>(d_img, d_x, d_y, width, height, numData);

	#ifdef TIMINGDEBUG
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elpsTime, start, stop));
	printf("\n\nElapsed time for creating build viz:      %9.6f ms \n", elpsTime);
	#endif

	return 0;
}

//Create filter visualization
int QuadTreeBuilder::createFilterViz()
{
	if ((numTestData < 0) || (numFilteredData < 0) || (numFilteredData > numTestData))
	{
		fprintf(stderr, "QuadTreeBuilder::createFilterViz(): filter parameters not configured correctly\n");
		return -1;
	}

	//Write Random data onto image buffer
	#ifdef TIMINGDEBUG
	checkCudaErrors(hipEventRecord(start, 0));
	#endif

	d_setBlackImag<<<gridDim, blockDim>>>(d_img, width, height);

	int blocksD = divUp(numNodes - numData, threads);
	std::cout << "BlocksD is " << blocksD << std::endl;
	d_drawCellInnerEdges<<<blocksD, threads>>>(d_img, d_index, d_x, d_y, d_rx, d_ry, width, height, numData, numNodes);

	//Write point last to avoid occulsion by lines (no alpha blending)
	d_writeData2Image<<<blocks, threads>>>(d_img, d_x, d_y, width, height, numTestData);
	
	blocksD = divUp(numFilteredData, threads);
	d_writeFilter2Image<<<blocksD, threads>>>(d_img, d_xf, d_yf, width, height, numFilteredData);

	#ifdef TIMINGDEBUG
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elpsTime, start, stop));
	printf("\n\nElapsed time for creating filter viz:     %9.6f ms \n", elpsTime);
	#endif

	return 0;
}

int QuadTreeBuilder::downloadData()
{
	//Copy back to host for checking    
	checkCudaErrors(hipMemcpy(h_x, d_x, dataSz, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_y, d_y, dataSz, hipMemcpyDeviceToHost));

	return 0;
}

int QuadTreeBuilder::downloadFilterData()
{
	//Copy back to host for checking    
	checkCudaErrors(hipMemcpy(h_xf, d_xf, numFilteredData*sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_yf, d_yf, numFilteredData*sizeof(float), hipMemcpyDeviceToHost));

	return 0;
}

int QuadTreeBuilder::resetData()
{
	if ((width<0) || (height<0))
	{
		fprintf(stderr, "QuadTreeBuilder::resetData(): width or height < 0, must initialize prior to data generation\n");
		return -1;
	}

	//Generate Random Data
	#ifdef TIMINGDEBUG
	checkCudaErrors(hipEventRecord(start, 0));
	#endif

	int seed = (int)time(0);

	generate_uniform2D_kernel<<<blocks, threads>>>(d_x, d_y, seed, width, height, numData);

	#ifdef TIMINGDEBUG
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elpsTime, start, stop));
	printf("\n\nElapsed time for Random Data Generation:  %9.6f ms \n", elpsTime);
	#endif

	return 0;
}

int QuadTreeBuilder::resetFilterData()
{
	if ((width<0) || (height<0))
	{
		fprintf(stderr, "QuadTreeBuilder::resetFilterData(): width or height < 0, must initialize prior to data generation\n");
		return -1;
	}

	//Generate Random Data
	#ifdef TIMINGDEBUG
	checkCudaErrors(hipEventRecord(start, 0));
	#endif

	int seed = (int)time(0);

	generate_uniform2Dfilter_kernel<<<blocks, threads>>>(d_x, d_y, d_score, seed, width, height, numTestData);

	#ifdef TIMINGDEBUG
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elpsTime, start, stop));
	printf("\n\nElapsed time make Random Filter Data:     %9.6f ms \n", elpsTime);
	#endif

	return 0;
}

//Resets arrays used in constructing the quad tree
void QuadTreeBuilder::ResetArrays(const int w, const int h)
{
	reset_arrays_kernel<<<blocks, threads>>>(d_mutex, d_x, d_y, d_rx, d_ry, d_child, d_index, d_left, d_right, d_bottom, d_top, w, h, numData, numNodes);
}
void QuadTreeBuilder::ResetFilterArrays(const int q, const int w, const int h)
{
	reset_filter_arrays_kernel<<<blocks, threads>>>(d_mutex, d_x, d_y, d_score, d_xf, d_yf, d_scoref, d_rx, d_ry, d_child, d_index,
												d_left, d_right, d_bottom, d_top, q, w, h, numData, numNodes);
}

//Builds a quad tree
void QuadTreeBuilder::BuildQuadTree()
{
	build_tree_kernel<<<blocks, threads>>>(d_x, d_y, d_rx, d_ry, d_child, d_index, d_left, d_right, d_bottom, d_top, numData, numNodes);
}

//Filter with quad tree
void QuadTreeBuilder::FilterQuadTree(const int d, const int q, const int f)
{
	#ifdef TIMINGDEBUG
	checkCudaErrors(hipEventRecord(start,0));
	#endif

	filter_tree_kernel<<<blocks, threads>>>(d_x, d_y, d_score, d_rx, d_ry, d_child, d_index, d_left, d_right, d_bottom, d_top, numData, numNodes, d, f);

	#ifdef TIMINGDEBUG
	checkCudaErrors(hipEventRecord(stop,0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elpsTime, start, stop));
	printf("\n\nElapsed time for QuadTree Filter:  %9.6f ms \n", elpsTime);

	checkCudaErrors(hipEventRecord(start,0));
	#endif

	pack_filtered_data_kernel<<<blocks, threads>>>(d_xf, d_yf, d_scoref, d_x, d_y, d_score, d_child, numData, d, q);

	#ifdef TIMINGDEBUG
	checkCudaErrors(hipEventRecord(stop,0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elpsTime, start, stop));
	printf("\n\nElapsed time for QuadTree Pack:  %9.6f ms \n", elpsTime);
	#endif
}

void QuadTreeBuilder::FilterQuadTreeDev(unsigned int* d, const int q, const int f)
{
	#ifdef TIMINGDEBUG
	checkCudaErrors(hipEventRecord(start,0));
	#endif

	filter_treeDev_kernel<<<blocks, threads>>>(d_x, d_y, d_score, d_rx, d_ry, d_child, d_index, d_left, d_right, d_bottom, d_top, numData, numNodes, d, f);

	#ifdef TIMINGDEBUG
	checkCudaErrors(hipEventRecord(stop,0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elpsTime, start, stop));
	printf("\n\nElapsed time for QuadTree Filter:  %9.6f ms \n", elpsTime);

	checkCudaErrors(hipEventRecord(start,0));
	#endif

	pack_filteredDev_data_kernel<<<blocks, threads>>>(d_xf, d_yf, d_scoref, d_x, d_y, d_score, d_child, numData, d, q);

	#ifdef TIMINGDEBUG
	checkCudaErrors(hipEventRecord(stop,0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elpsTime, start, stop));
	printf("\n\nElapsed time for QuadTree Pack:  %9.6f ms \n", elpsTime);
	#endif
}

}	//namespace QuadTree
